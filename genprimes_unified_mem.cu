/* 
 *  Last name: Gupta  
 *  First name: Vaibhav
 *  Net ID: vvg239
 * 
 */

 
#include <hip/hip_runtime.h>
#include <stdlib.h>
 #include <stdio.h>
 #include <stdbool.h>
 #include <string.h>
 #include <time.h> 
 
 void seq_gen_primes(int);
 __global__ void gen_primes(bool*, unsigned int, int);
 void gpu_gen_primes(unsigned int);
 
 int main(int argc, char * argv[])
 {
   int N;
   // to measure time taken by a specific part of the code 
   double time_taken;
   clock_t start, end;

   if(argc == 2) 
   {
     N = atoi(argv[1]);
   }
   else
   {
     printf("Please give a value for N\n");
   }

   start = clock();
   gpu_gen_primes(N);
   end = clock();

   time_taken = ((double)(end - start))/ CLOCKS_PER_SEC;
   printf("Time taken for %s is %lf\n","GPU", time_taken);
 }

 /******************** The GPU parallel version **************/
 void  gpu_gen_primes(unsigned int N)
{
   FILE * fPtr;
   char fileName[15];
   sprintf(fileName, "%d", N);
   strcat(fileName, ".txt");
   fPtr = fopen(fileName, "w");

   int size = (N+1) * sizeof(bool);
   bool * n_series;
   hipMallocManaged((void**) &n_series, size);
   int last_divisor = (N+1)/2;
   for(int divisor = 2; divisor < last_divisor; divisor++) {
     if(n_series[divisor]) continue;
     int num_threads = 512;
     int num_blocks = N/(divisor*num_threads) + 1;
     gen_primes<<<num_blocks, num_threads>>>(n_series, N, divisor);
     hipDeviceSynchronize();
     hipError_t error = hipGetLastError();
     if(error != hipSuccess) {
       printf("CUDA error %s \n", hipGetErrorString(error));
       break;
     }
   }

   int i;
   for(i = 2; i < N+1; i++)
   {
     if(!n_series[i]) {
       fprintf(fPtr, "%d ", i);
     }
   }
   fprintf(fPtr, "\n");
   hipFree(n_series);
}

 __global__
void gen_primes(bool* n_series, unsigned int N, int divisor) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int e = divisor*(i+2);
  if(e <= N) {
    n_series[e] = true;
  }
}
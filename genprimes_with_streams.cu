/* 
 *  Last name: Gupta  
 *  First name: Vaibhav
 *  Net ID: vvg239
 * 
 */

 
#include <hip/hip_runtime.h>
#include <stdlib.h>
 #include <stdio.h>
 #include <stdbool.h>
 #include <string.h>
 #include <time.h> 
 
 void seq_gen_primes(int);
 __global__ void remove_for_divisor(bool*, unsigned int, int);
 __global__ void remove_all(bool*, unsigned int, unsigned int);
 void gpu_gen_primes(unsigned int);
 
 int main(int argc, char * argv[])
 {
   int N;
   // to measure time taken by a specific part of the code 
   double time_taken;
   clock_t start, end;

   if(argc == 2) 
   {
     N = atoi(argv[1]);
   }
   else
   {
     printf("Please give a value for N\n");
   }

   start = clock();
   gpu_gen_primes(N);
   end = clock();

   time_taken = ((double)(end - start))/ CLOCKS_PER_SEC;
   printf("Time taken for %s is %lf\n","GPU", time_taken);
 }

 /******************** The GPU parallel version **************/
 void  gpu_gen_primes(unsigned int N)
{
   //File Stream Initialization
   FILE * fPtr;
   char fileName[15];
   sprintf(fileName, "%d", N);
   strcat(fileName, ".txt");
   fPtr = fopen(fileName, "w");
   
   double time_taken;
   clock_t start, end;
   start = clock();
   //CUDA Memory Allocation
   int size = (N+1) * sizeof(bool);
   bool * d_primes;
   hipMalloc(&d_primes, size);

   //Configuring CUDA Kernels
   unsigned int last_divisor = sqrt(N);
   int num_threads = 1024;
   int num_blocks = last_divisor/(num_threads) + 1;
   int num_threads_for_two = 1024;
   int num_blocks_for_two = N/(num_threads_for_two*2) + 1;

   //Call kernels in Streams
   hipStream_t stream[4]; 
   int ds[4] = {2,3,5,7};
   for (int i = 0; i < 4; i++) {
    hipStreamCreate(&stream[i]); 
    remove_for_divisor<<<N/(num_threads_for_two*ds[i]) + 1, num_threads_for_two,0,stream[i]>>>(d_primes, N, ds[i]);
   }
   remove_all<<<num_blocks, num_threads, 0, stream[0]>>>(d_primes, N, last_divisor);
   hipDeviceSynchronize();
   hipError_t error = hipGetLastError();
   if(error != hipSuccess) {
     printf("CUDA error %s \n", hipGetErrorString(error));
   }
   end = clock();
   time_taken = ((double)(end - start))/ CLOCKS_PER_SEC;
   printf("Time taken without print statements for %s is %lf\n","GPU", time_taken);

   //Copy CUDA Memory and Print in File
   bool * primes;
   primes = (bool *)calloc(N, sizeof(bool));
   hipMemcpy(primes, d_primes, size, hipMemcpyDeviceToHost);
   hipFree(d_primes);
   int i;
   for(i = 2; i < N+1; i++)
   {
     if(!primes[i]) {
       fprintf(fPtr, "%d ", i);
     }
   }

}

 __global__
void remove_for_divisor(bool* n_series, unsigned int N, int divisor) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int e = divisor*(i+2);
  if(e <= N) {
    n_series[e] = true;
  }
}

__global__
void remove_all(bool* n_series, unsigned int N, unsigned int max_divisor) {

  int divisor = blockIdx.x * blockDim.x + threadIdx.x + 3;

  // this might initialize for some divisors (like 9) that are not prime but this 
  // still gives better performance, than waiting for 3 to finish and then executing 5,7 and 11.
  if (divisor <= max_divisor && n_series[divisor] == false) {

    // start marking off from (divisor)^2
    for (int j = divisor * divisor; j <= N; j += divisor) {
      n_series[j] = true;
    }

  }
}